#include "hip/hip_runtime.h"
﻿#include <iostream>
#include "Shape.h"
#include "Texture.h"
#include "ShaderClass.h"
#include "Camera.h"

#include "hip/hip_runtime.h"
#include ""



#include <stdio.h>

using namespace std;



const int RT_WIDTH = 1280;
const int RT_HEIGHT = 720;
const float ZNEAR = 0.1f;
const float ZFAR = 100.0f;

class SphInfo {
public:
    SphInfo(glm::vec3 pos, glm::vec4 alb, float r, int t, float rh ) : 
        mSphPos(pos), mAlbedo(alb), mRad(r), mType(t), mRoughness(rh) {}

    glm::vec3 mSphPos;
    glm::vec4 mAlbedo;
    float mRad;
    int mType;
    float mRoughness;
};

vector<SphInfo> gSphInfos;

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}



int main()
{
#pragma region OpenGL_code_init_window_creation_vertices_etc
    glfwInit();
    glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 3);
    glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 3);
    glfwWindowHint(GLFW_OPENGL_PROFILE, GLFW_OPENGL_CORE_PROFILE);


    GLfloat vertices[] = {
        -1.0f, -1.0f, 0.0f,  1.0f, 0.0f, 0.0f, 0.0f, 0.0f,
        +1.0f, -1.0f, 0.0f,  0.0f, 1.0f, 0.0f, 1.0f, 0.0f,
        +1.0f, +1.0f, 0.0f,  0.0f, 0.0f, 1.0f, 1.0f, 1.0f,
        -1.0f, +1.0f, 0.0f,  1.0f, 1.0f, 1.0f, 0.0f, 1.0f,
    };

    GLuint indices[] = { 0,1,2, 0,2,3};


    GLFWwindow* window = glfwCreateWindow(RT_WIDTH, RT_HEIGHT, "RTWin", NULL, NULL);

    if (window == NULL) { cout << "Window creation failed" << endl; glfwTerminate(); return -1; }

    glfwMakeContextCurrent(window);
    gladLoadGL();

#pragma endregion

    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

#pragma region OpenGL_code_viewport_clear_shader_while_poll_clean_up

    

    glm::vec4 botSkyCol = glm::vec4(1, 1, 1, 1);
    glm::vec4 topSkyCol = glm::vec4(0.5, 0.7, 1, 1);

    gSphInfos.push_back(SphInfo(glm::vec3(0, 0, -1),            glm::vec4(0.1, 0.2, 0.5, 1), 0.5, 0, 1.0));
    gSphInfos.push_back(SphInfo(glm::vec3(0, -100.5, -1),       glm::vec4(0.8, 0.8, 0.0, 1), 100, 0, 1.0));
    gSphInfos.push_back(SphInfo(glm::vec3(1, 0, -1),            glm::vec4(0.8, 0.6, 0.2, 1), 0.5, 1, 0.8));


    Shader shdrProgFalseSky("FalseSkyVS.h", "FalseSkyFS.h");
    Shader lambertShdr("lambertVS.h", "lambertFS.h");
    Shader shinyShdr("shinyVS.h", "shinyFS.h");

    Shape triShp(vertices, sizeof(vertices), indices, sizeof(indices));
    Sphere sphShp; sphShp.Create();

    Texture tex0("brick.png");

    float rotation = 0.0f;
    double prevTime = glfwGetTime();
    double currTime = prevTime;

    glm::mat4 model = glm::mat4(1.0f);

    Camera cam(RT_WIDTH, RT_HEIGHT, glm::vec3(0.0f, 0.5f, 2.0f));
    cam.ViewMatrix(); cam.ProjMatrix(45.0f, ZNEAR, ZFAR);
    
    glViewport(0, 0, RT_WIDTH, RT_HEIGHT);

    glClearColor(0.077f, 0.13f, 0.17f, 1.0f);
    glClear(GL_COLOR_BUFFER_BIT);
    glfwSwapBuffers(window);

    glEnable(GL_DEPTH_TEST);

    while (!glfwWindowShouldClose(window)) {

        currTime = glfwGetTime();

        float dTime = (currTime - prevTime);
        prevTime = currTime;


        glClearColor(0.077f, 0.13f, 0.17f, 1.0f);
        glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

        shdrProgFalseSky.Activate();
        shdrProgFalseSky.SetVec4("botSkyColor", botSkyCol);
        shdrProgFalseSky.SetVec4("topSkyColor", topSkyCol);
        //tex0.ActivateAndBind();
        //shdrProgFalseSky.SetInt("texture0", 0);

        triShp.Draw();

        //draw lambertspheres
        for (int i = 0; i < gSphInfos.size(); i++) {

            model = glm::mat4(1.0f);
            float sc = gSphInfos[i].mRad * 2.0f;
            model = glm::translate(model, gSphInfos[i].mSphPos);
            model = glm::scale(model, glm::vec3(sc,sc,sc));

            if (gSphInfos[i].mType == 0) {

                lambertShdr.Activate();
                lambertShdr.SetMat4("model", model);
                lambertShdr.SetMat4("view", cam.mView);
                lambertShdr.SetMat4("proj", cam.mProj);
                lambertShdr.SetVec4("botSkyColor", botSkyCol);
                lambertShdr.SetVec4("topSkyColor", topSkyCol);
                lambertShdr.SetVec4("albedo", gSphInfos[i].mAlbedo);

            }
            else if (gSphInfos[i].mType == 1) {

                shinyShdr.Activate();
                shinyShdr.SetMat4("model", model);
                shinyShdr.SetMat4("view", cam.mView);
                shinyShdr.SetMat4("proj", cam.mProj);
                shinyShdr.SetVec3("camPosW", cam.mPos);
                shinyShdr.SetVec4("botSkyColor", botSkyCol);
                shinyShdr.SetVec4("topSkyColor", topSkyCol);
                shinyShdr.SetVec4("albedo", gSphInfos[i].mAlbedo);
                shinyShdr.SetFloat("roughness", gSphInfos[i].mRoughness);

            }

            sphShp.Draw();
        }
 


        glfwSwapBuffers(window);


        glfwPollEvents();
        cam.Inputs(window,dTime);
        cam.ViewMatrix();
    }

    triShp.Delete();
    shdrProgFalseSky.Delete();
    tex0.Delete();

    glfwDestroyWindow(window);
    glfwTerminate();

#pragma endregion

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
